#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>

#define BLOCKDIMX 128
#define GRIDDIMX 100


__global__ void add(int* a, int* b, int* c){
    int idx = blockIdx.x * BLOCKDIMX + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}


int main()
{
    int NB_BLOCK = GRIDDIMX;
    int SIZE = 1*NB_BLOCK  ;
    int h_a[SIZE], h_b[SIZE], h_c[SIZE];
    int *d_a, *d_b, *d_c;
    int *d_d, *d_e, *d_f;
    // int* h_c = (int*)malloc(sizeof(int)*SIZE);
    for(int i = 0; i < SIZE; ++i){
        h_a[i] = i;
        h_b[i] = SIZE-i;
    }

    hipStream_t stream1;
    hipStream_t stream2;

    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);


    hipMalloc((void**)&d_a, SIZE * sizeof(int));
    hipMalloc((void**)&d_b, SIZE * sizeof(int));
    hipMalloc((void**)&d_c, SIZE * sizeof(int));

    hipMalloc((void**)&d_d, SIZE * sizeof(int));
    hipMalloc((void**)&d_e, SIZE * sizeof(int));
    hipMalloc((void**)&d_f, SIZE * sizeof(int));

    hipMemcpyAsync(d_a, h_a, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_b, h_b, SIZE*sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_d, h_a, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_e, h_b, SIZE*sizeof(int), hipMemcpyHostToDevice);

    add<<<NB_BLOCK, BLOCKDIMX, 0, stream1>>>(d_a, d_b, d_c);
    add<<<NB_BLOCK, BLOCKDIMX, 0, stream2>>>(d_d, d_e, d_f);
    // add<<<1, 1>>>(0,0,0);

    hipMemcpy(h_c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_e);
    hipFree(d_f);
    int i = 0;
    for(i = 0; i < SIZE; ++i){
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl << i << std::endl;

    return 0;
}
